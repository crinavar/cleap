#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//                                                                           	//
//	cleap                                                                   //
//	A library for handling / processing / rendering 3D meshes.	        //
//                                                                           	//
//////////////////////////////////////////////////////////////////////////////////
//										//
//	Copyright © 2011 Cristobal A. Navarro.					//
//										//	
//	This file is part of cleap.						//
//	cleap is free software: you can redistribute it and/or modify		//
//	it under the terms of the GNU General Public License as published by	//
//	the Free Software Foundation, either version 3 of the License, or	//
//	(at your option) any later version.					//
//										//
//	cleap is distributed in the hope that it will be useful,		//
//	but WITHOUT ANY WARRANTY; without even the implied warranty of		//
//	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the	    	//
//	GNU General Public License for more details.				//
//										//
//	You should have received a copy of the GNU General Public License	//
//	along with cleap.  If not, see <http://www.gnu.org/licenses/>. 		//
//										//
//////////////////////////////////////////////////////////////////////////////////



// public headers
#include "cleap_private.h"
#include "cleap_config.h"

// important cuda runtime headers
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// kernel headers
#include "cleap_kernel_utils.cu"
#include "cleap_kernel_normalize_normals.cu"
#include "cleap_kernel_delaunay_transformation.cu"
#include "cleap_kernel_paint_mesh.cu"
#include "cleap_kernel_fix_inverted_triangles.cu"

// context creation header for opengl
// linux
#include "cleap_glx_context.cu"

// default blocksize
int CLEAP_CUDA_BLOCKSIZE = 256;

// timer structures
struct timeval t_ini, t_fin;

// cleap author
char CLEAP_AUTHOR[] = "Cristobal A. Navarro";

// cuda textures
texture<GLuint, 1, hipReadModeElementType> tex_triangles;
texture<int, 1, hipReadModeElementType> tex_edges;

int cleap_mesh_is_wireframe(_cleap_mesh *m){
	return m->wireframe;
}
int cleap_mesh_is_solid(_cleap_mesh *m){
	return m->solid;
}
void cleap_mesh_set_wireframe(_cleap_mesh *m, int w){
	m->wireframe = w;
}
void cleap_mesh_set_solid(_cleap_mesh *m, int s){
	m->solid = s;
}

float cleap_get_bsphere_r(_cleap_mesh *m){

        float view_diamx = m->max_coords.x - m->min_coords.x;
        float view_diamy = m->max_coords.y - m->min_coords.y;
        float view_diamz = m->max_coords.z - m->min_coords.z;
	return 0.5f*sqrt(powf(view_diamx, 2) + powf(view_diamz, 2) + powf(view_diamy, 2));
}
float cleap_get_bsphere_x(_cleap_mesh *m){

	return	0.5f*(m->max_coords.x + m->min_coords.x);
}
float cleap_get_bsphere_y(_cleap_mesh *m){

	return	0.5f*(m->max_coords.y + m->min_coords.y);
}
float cleap_get_bsphere_z(_cleap_mesh *m){

	return	0.5f*(m->max_coords.z + m->min_coords.z);
}


CLEAP_RESULT cleap_init(){

	_cleap_print_splash();
	_cleap_init_cuda();

	return CLEAP_SUCCESS;

}

CLEAP_RESULT cleap_init_no_render(){

	//_cleap_print_splash();
	_cleap_create_glx_context();
	_cleap_init_glew();
	_cleap_init_cuda();

	return CLEAP_SUCCESS;
}

CLEAP_RESULT cleap_end(){
	_cleap_destroy_glx_context();
	return CLEAP_SUCCESS;
}

int cleap_get_vertex_count(_cleap_mesh *hm){
	return hm->vertex_count;
}

int cleap_get_edge_count(_cleap_mesh *hm){
	return hm->edge_count;
}

int cleap_get_face_count(_cleap_mesh *hm){
	return hm->face_count;
}

_cleap_mesh* cleap_load_mesh(const char* filename){

	_cleap_mesh *m = new _cleap_mesh();	// create mew mesh
	_cleap_host_load_mesh(m, filename);	// load host part
	_cleap_device_load_mesh(m);		// load device part

	return m;
}

CLEAP_RESULT cleap_paint_mesh(_cleap_mesh *m, GLfloat r, GLfloat g, GLfloat b, GLfloat a ){

	//printf("CLEAP::kernel::paint_mesh::");
	size_t bytes;
	float4 *dptr;
	int vcount = cleap_get_vertex_count(m);
	cleap_device_mesh *dm = m->dm;
	hipGraphicsMapResources(1, &dm->vbo_c_cuda, 0);
	hipGraphicsResourceGetMappedPointer((void **)&dptr, &bytes, dm->vbo_c_cuda);

	dim3 dimBlock(CLEAP_CUDA_BLOCKSIZE);
	dim3 dimGrid((vcount+CLEAP_CUDA_BLOCKSIZE) / dimBlock.x);
	hipDeviceSynchronize();
	cleap_kernel_paint_mesh<<< dimGrid, dimBlock >>>(dptr, vcount, r, g, b, a);
	hipDeviceSynchronize();
	// unmap buffer object
	hipGraphicsUnmapResources(1, &dm->vbo_c_cuda, 0);
	//printf("ok\n");

	return CLEAP_SUCCESS;
}

CLEAP_RESULT cleap_render_mesh(_cleap_mesh *m){

	if(m->status == CLEAP_SUCCESS && m->dm->status == CLEAP_SUCCESS){
	  	glEnable (GL_POLYGON_OFFSET_FILL); 	//Necesario para permitir dibujar 2 poligonos
    		glPolygonOffset (1.0, 1.0); 		//coplanares (Wireframe y poligono solido)
		glEnable(GL_BLEND);
		glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, m->dm->eab);

		//! position vectors
		glBindBuffer(GL_ARRAY_BUFFER, m->dm->vbo_v);
		glEnableClientState(GL_VERTEX_ARRAY);
		glVertexPointer(3,      GL_FLOAT, 4*sizeof(float), 0);
		//! normal vectors
		glBindBuffer(GL_ARRAY_BUFFER, m->dm->vbo_n);
		glEnableClientState(GL_NORMAL_ARRAY);
		glNormalPointer(        GL_FLOAT, 4*sizeof(float), 0);
		//! color vectors
		glBindBuffer(GL_ARRAY_BUFFER, m->dm->vbo_c);
		glEnableClientState(GL_COLOR_ARRAY);
		glColorPointer(4,       GL_FLOAT, 4*sizeof(float), 0);

		if (m->solid){
			glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
			glDrawElements(GL_TRIANGLES, cleap_get_face_count(m)*3, GL_UNSIGNED_INT, BUFFER_OFFSET(0));
		}
		if (m->wireframe){
			glDisableClientState(GL_COLOR_ARRAY);
			glColor3f(0.0f, 0.0f, 1.0f);
			glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
			glDrawElements(GL_TRIANGLES, cleap_get_face_count(m)*3, GL_UNSIGNED_INT, BUFFER_OFFSET(0));
		}
		glBindBuffer(GL_ARRAY_BUFFER, 0);
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
		glDisableClientState(GL_COLOR_ARRAY);
		glDisable(GL_POLYGON_OFFSET_FILL);
		glDisable(GL_BLEND);
	}
	return CLEAP_SUCCESS;
	
}

CLEAP_RESULT cleap_sync_mesh(_cleap_mesh *m){

	float4 *d_vbo_v, *d_vbo_n, *d_vbo_c;
	GLuint *d_eab;

	size_t num_bytes=0;
	int mem_size_vbo = cleap_get_vertex_count(m)*sizeof(float4);
	int mem_size_eab = 3*cleap_get_face_count(m)*sizeof(GLuint);
	int mem_size_edges = sizeof(int2)*cleap_get_edge_count(m);

	hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->vbo_n_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->vbo_c_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);

	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &num_bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_n, &num_bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_c, &num_bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &num_bytes, m->dm->eab_cuda);

	hipMemcpy( m->vnc_data.v, d_vbo_v, mem_size_vbo, hipMemcpyDeviceToHost );
	hipMemcpy( m->vnc_data.n, d_vbo_n, mem_size_vbo, hipMemcpyDeviceToHost );
	hipMemcpy( m->vnc_data.c, d_vbo_c, mem_size_vbo, hipMemcpyDeviceToHost );
	hipMemcpy( m->triangles, d_eab, mem_size_eab, hipMemcpyDeviceToHost );

	hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->vbo_n_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->vbo_c_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);

	hipMemcpy( m->edge_data.n, m->dm->d_edges_n, mem_size_edges, hipMemcpyDeviceToHost );
	hipMemcpy( m->edge_data.a, m->dm->d_edges_a, mem_size_edges, hipMemcpyDeviceToHost );
	hipMemcpy( m->edge_data.b, m->dm->d_edges_b, mem_size_edges, hipMemcpyDeviceToHost );
	hipMemcpy( m->edge_data.op, m->dm->d_edges_op, mem_size_edges, hipMemcpyDeviceToHost );

	return CLEAP_SUCCESS;

}

void cleap_print_mesh( _cleap_mesh *m ){

	cleap_sync_mesh(m);
	float4 *d_vbo_v, *d_vbo_n, *d_vbo_c;
	GLuint *d_eab;
	float4 *h_vbo_v, *h_vbo_n, *h_vbo_c;
	GLuint *h_eab;

	h_vbo_v = (float4*)malloc(cleap_get_vertex_count(m)*sizeof(float4));
	h_vbo_n = (float4*)malloc(cleap_get_vertex_count(m)*sizeof(float4));
	h_vbo_c = (float4*)malloc(cleap_get_vertex_count(m)*sizeof(float4));
	h_eab = (GLuint*)malloc(3*cleap_get_face_count(m)*sizeof(GLuint));

	size_t num_bytes=0;
	int mem_size_vbo = cleap_get_vertex_count(m)*sizeof(float4);
	int mem_size_eab = 3*cleap_get_face_count(m)*sizeof(GLuint);

	hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->vbo_n_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->vbo_c_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);

	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &num_bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_n, &num_bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_c, &num_bytes, m->dm->vbo_v_cuda);

	hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &num_bytes, m->dm->eab_cuda);

	hipMemcpy( h_vbo_v, d_vbo_v, mem_size_vbo, hipMemcpyDeviceToHost );
	hipMemcpy( h_vbo_n, d_vbo_n, mem_size_vbo, hipMemcpyDeviceToHost );
	hipMemcpy( h_vbo_c, d_vbo_c, mem_size_vbo, hipMemcpyDeviceToHost );

	hipMemcpy( h_eab, d_eab, mem_size_eab, hipMemcpyDeviceToHost );

	hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->vbo_n_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->vbo_c_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);

	for(int i=0; i<cleap_get_vertex_count(m); i++){
		printf("mesh_data[%i] = (%f, %f, %f)  w=%f\n", i, h_vbo_v[i].x, h_vbo_v[i].y, h_vbo_v[i].z, h_vbo_v[i].w);
	}
	for(int i=0; i<cleap_get_face_count(m); i++){
		printf("T[%i] = (%i, %i, %i)\n", i, h_eab[3*i], h_eab[3*i+1], h_eab[3*i+2]);
	}

	for( int i=0; i<cleap_get_edge_count(m); i++ ){
	    printf("edge[%i]:\n", i);
	    printf("\tn = (%i, %i)\t", m->edge_data.n[i].x, m->edge_data.n[i].y);
	    printf("a = (%i, %i)\t", m->edge_data.a[i].x, m->edge_data.a[i].y);
	    printf("b = (%i, %i)\n", m->edge_data.b[i].x, m->edge_data.b[i].y);
	}
}

CLEAP_RESULT cleap_fix_inverted_triangles(cleap_mesh* m){
    return cleap_fix_inverted_triangles_mode(m,CLEAP_MODE_2D);
}

CLEAP_RESULT cleap_fix_inverted_triangles_mode(_cleap_mesh *m, int mode){
    //printf("CLEAP::fix_inverted_triangles%id\n", mode);
    float4 *d_vbo_v;
    GLuint *d_eab;
    size_t bytes=0;
    int *h_listo, it=0;
    // Map resources
    hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
    hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);
    hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &bytes, m->dm->vbo_v_cuda);
    hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &bytes, m->dm->eab_cuda);
    // TEXTURE
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<GLuint>();
    hipBindTexture(0, tex_triangles, d_eab, channelDesc, cleap_get_face_count(m)*3*sizeof(GLuint));
    int block_size = CLEAP_CUDA_BLOCKSIZE;
    dim3 dimBlock(block_size);
    dim3 dimGrid((cleap_get_edge_count(m)+block_size-1) / dimBlock.x);
    dim3 dimBlockInit(block_size);
    dim3 dimGridInit((cleap_get_face_count(m)+block_size-1) / dimBlock.x);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // if C.C is 1.2 or higher, then use zero-copy for the flag
    if( (deviceProp.major == 1 && deviceProp.minor >= 2) || (deviceProp.major >= 2) ){
        //printf("CLEAP::device::gpu::%s\n", deviceProp.name );
        //printf("CLEAP::device_property::canMapHostMemory = %i\n", deviceProp.canMapHostMemory);
        hipHostAlloc((void **)&h_listo, sizeof(int), hipHostMallocMapped);
        h_listo[0] = 0;
        hipHostGetDevicePointer((void **)&m->dm->d_listo, (void *)h_listo, 0);
        _cleap_start_timer();

        while( !h_listo[0] ){
            h_listo[0] = 1;
            hipDeviceSynchronize();
            _cleap_init_device_dual_arrays_int(m->dm->d_trirel, m->dm->d_trireservs, cleap_get_face_count(m), -1, dimBlockInit, dimGridInit); //demora el orden de 10^-5 secs
            hipDeviceSynchronize();
            if( mode == CLEAP_MODE_2D )
                correctTrianglesKernel<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
            else
                cleap_kernel_triangle_fix_3d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs); // NOT WORKING AS INTENDED

            hipDeviceSynchronize();
            if( h_listo[0] ){break;}
            cleap_kernel_repair<<< dimGrid, dimBlock >>>(d_eab, m->dm->d_trirel, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m)); //update
            it++;
        }
    }
        // else use memcpy transfers
    else{
        //! ZERO COPY = OFF
        //printf("CLEAP::device::gpu::%s\n", deviceProp.name );
        h_listo = (int*)malloc(sizeof(int));
        h_listo[0] = 0;
        hipMalloc( (void**) &m->dm->d_listo , sizeof(int) );
        //listo es una variable que indica cuando el algoritmo ha finalizado. cuanto listo = 1 entonces todos los edges son delaunay.
        _cleap_start_timer();
        while( !h_listo[0] ){

            h_listo[0] = 1;
            hipMemcpy( m->dm->d_listo, h_listo, sizeof(int), hipMemcpyHostToDevice );
            _cleap_init_device_dual_arrays_int(m->dm->d_trirel, m->dm->d_trireservs, cleap_get_face_count(m), -1, dimBlockInit, dimGridInit); //demora el orden de 10^-5 secs
            if( mode == CLEAP_MODE_2D )
                correctTrianglesKernel<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
            else
                cleap_kernel_triangle_fix_3d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs); // NOT WORKING AS INTENDED

            hipDeviceSynchronize();
            hipMemcpy( h_listo, m->dm->d_listo, sizeof(int), hipMemcpyDeviceToHost );
            if( h_listo[0] ){
                break;
            }
            cleap_kernel_repair<<< dimGrid, dimBlock >>>(d_eab, m->dm->d_trirel, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m)); //update
            it++;
        }
        hipFree(m->dm->d_listo);
    }
    //printf("computed in %.5g[s] (%i iterations)\n", _cleap_stop_timer(), it );
    //printf("%.6f\n", _cleap_stop_timer());
    //!Unbind Texture
    hipUnbindTexture(tex_triangles);
    // unmap buffer object
    hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
    hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);
    hipHostFree(h_listo);

    return CLEAP_SUCCESS;

}

CLEAP_RESULT cleap_random_move_points(cleap_mesh* m, float maxDisturb){
    //printf("CLEAP::delaunay_transformation_%id\n", mode);
    float4 *d_vbo_v;
    GLuint *d_eab;
    size_t bytes=0;
    int it=0;
    // Map resources
    hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
    hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &bytes, m->dm->vbo_v_cuda);

    int block_size = CLEAP_CUDA_BLOCKSIZE;
    dim3 dimBlock(block_size);
    dim3 dimGrid((cleap_get_vertex_count(m)+block_size-1) / dimBlock.x);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    //printf("CLEAP::device::gpu::%s\n", deviceProp.name );
    //printf("CLEAP::device_property::canMapHostMemory = %i\n", deviceProp.canMapHostMemory);
    _cleap_start_timer();

    float displacements[2*cleap_get_vertex_count(m)];
    for(int i=0;i<2*cleap_get_vertex_count(m);i++){
        displacements[i] = ((float)2*cleap_get_vertex_count(m)-i)/cleap_get_vertex_count(m)/2 * maxDisturb;
    }
    displacements[0]=0;
    displacements[1]=0;
    displacements[2]=maxDisturb;
    displacements[3]=0;
    displacements[4]=maxDisturb;
    displacements[5]=maxDisturb;
    displacements[6]=0;
    displacements[7]=maxDisturb;

    float* d_displacements;
    hipMalloc((void**)&d_displacements,2*cleap_get_vertex_count(m)*sizeof(float));
    hipMemcpy(d_displacements,displacements,2*cleap_get_vertex_count(m)*sizeof(float),hipMemcpyHostToDevice);

    cleap_random_move_points_kernel<256><<< dimGrid, dimBlock >>>(d_vbo_v, (float2*)d_displacements, cleap_get_vertex_count(m));
    hipFree(d_displacements);

    // unmap buffer object
    hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
    hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);
    return CLEAP_SUCCESS;
}

CLEAP_RESULT cleap_delaunay_transformation(_cleap_mesh *m, int mode){

	//printf("CLEAP::delaunay_transformation_%id\n", mode);
	float4 *d_vbo_v;
	GLuint *d_eab;
	size_t bytes=0;
	int *h_listo, it=0;
	// Map resources
	hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &bytes, m->dm->eab_cuda);
	// TEXTURE
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<GLuint>();
	hipBindTexture(0, tex_triangles, d_eab, channelDesc, cleap_get_face_count(m)*3*sizeof(GLuint));
	int block_size = CLEAP_CUDA_BLOCKSIZE;
	dim3 dimBlock(block_size);
	dim3 dimGrid((cleap_get_edge_count(m)+block_size-1) / dimBlock.x);
	dim3 dimBlockInit(block_size);
	dim3 dimGridInit((cleap_get_face_count(m)+block_size-1) / dimBlock.x);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	// if C.C is 1.2 or higher, then use zero-copy for the flag
	if( (deviceProp.major == 1 && deviceProp.minor >= 2) || (deviceProp.major >= 2) ){
		//printf("CLEAP::device::gpu::%s\n", deviceProp.name );
		//printf("CLEAP::device_property::canMapHostMemory = %i\n", deviceProp.canMapHostMemory);
		hipHostAlloc((void **)&h_listo, sizeof(int), hipHostMallocMapped);
		h_listo[0] = 0;
		hipHostGetDevicePointer((void **)&m->dm->d_listo, (void *)h_listo, 0);
		_cleap_start_timer();
		while( !h_listo[0] ){
			h_listo[0] = 1;
			hipDeviceSynchronize();
			_cleap_init_device_dual_arrays_int(m->dm->d_trirel, m->dm->d_trireservs, cleap_get_face_count(m), -1, dimBlockInit, dimGridInit); //demora el orden de 10^-5 secs
			hipDeviceSynchronize();
			if( mode == CLEAP_MODE_2D )
				cleap_kernel_exclusion_processing_2d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
			else 
				cleap_kernel_exclusion_processing_3d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
			
			hipDeviceSynchronize();
			if( h_listo[0] ){break;}
			cleap_kernel_repair<<< dimGrid, dimBlock >>>(d_eab, m->dm->d_trirel, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m)); //update
			it++;
		}
	}
	// else use memcpy transfers
	else{
		//! ZERO COPY = OFF
		//printf("CLEAP::device::gpu::%s\n", deviceProp.name );
		h_listo = (int*)malloc(sizeof(int));
		h_listo[0] = 0;
		hipMalloc( (void**) &m->dm->d_listo , sizeof(int) );
		//listo es una variable que indica cuando el algoritmo ha finalizado. cuanto listo = 1 entonces todos los edges son delaunay.
		_cleap_start_timer();
		while( !h_listo[0] ){

			h_listo[0] = 1;
			hipMemcpy( m->dm->d_listo, h_listo, sizeof(int), hipMemcpyHostToDevice );
			_cleap_init_device_dual_arrays_int(m->dm->d_trirel, m->dm->d_trireservs, cleap_get_face_count(m), -1, dimBlockInit, dimGridInit); //demora el orden de 10^-5 secs
			if( mode == CLEAP_MODE_2D )
				cleap_kernel_exclusion_processing_2d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
			else 
				cleap_kernel_exclusion_processing_3d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
			
			hipDeviceSynchronize();
			hipMemcpy( h_listo, m->dm->d_listo, sizeof(int), hipMemcpyDeviceToHost );
			if( h_listo[0] ){
				break;
			}
			cleap_kernel_repair<<< dimGrid, dimBlock >>>(d_eab, m->dm->d_trirel, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m)); //update
			it++;
		}
		hipFree(m->dm->d_listo);
	}
	//printf("computed in %.5g[s] (%i iterations)\n", _cleap_stop_timer(), it );
	//printf("%.6f\n", _cleap_stop_timer());
	//!Unbind Texture
	hipUnbindTexture(tex_triangles);
	// unmap buffer object
	hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);
	hipHostFree(h_listo);

	return CLEAP_SUCCESS;

}


int cleap_delaunay_transformation_interactive(_cleap_mesh *m, int mode){

	float4 *d_vbo_v;
	GLuint *d_eab;
	size_t bytes=0;
	int *h_listo, it=0, *flips;

	hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &bytes, m->dm->eab_cuda);

	// TEXTURE
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<GLuint>();
	hipBindTexture(0, tex_triangles, d_eab, channelDesc, cleap_get_face_count(m)*3*sizeof(GLuint));
	int block_size = CLEAP_CUDA_BLOCKSIZE;
	dim3 dimBlock(block_size);
	dim3 dimGrid((cleap_get_edge_count(m)+block_size-1) / dimBlock.x);
	dim3 dimBlockInit(block_size);
	dim3 dimGridInit((cleap_get_face_count(m)+block_size-1) / dimBlock.x);
	hipHostAlloc((void **)&h_listo, sizeof(int), hipHostMallocMapped);
	hipHostAlloc((void **)&flips, sizeof(int), hipHostMallocMapped);
	h_listo[0] = 0;
	flips[0] = 0;
	int *dflips;
	hipHostGetDevicePointer((void **)&m->dm->d_listo, (void *)h_listo, 0);
	hipHostGetDevicePointer((void **)&dflips, (void *)flips, 0);		
	_cleap_start_timer();
	// compute iteration
	h_listo[0] = 1;
	hipDeviceSynchronize();
	_cleap_init_device_dual_arrays_int(m->dm->d_trirel, m->dm->d_trireservs, cleap_get_face_count(m), -1, dimBlockInit, dimGridInit); //demora el orden de 10^-5 secs
	hipDeviceSynchronize();
	if( mode == CLEAP_MODE_2D )
		cleap_kernel_exclusion_processing_2d_debug<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs, dflips);
	else 
		cleap_kernel_exclusion_processing_3d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
	
	hipDeviceSynchronize();
	if( h_listo[0] ){
		hipUnbindTexture(tex_triangles);
		// unmap buffer object
		hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
		hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);
		hipHostFree(h_listo);
		return 0;
	}
	cleap_kernel_repair<<< dimGrid, dimBlock >>>(d_eab, m->dm->d_trirel, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m)); //update
	it++;
	//printf("CLEAP::delaunay_transformation_%id:: Iteration computed in %.5g[s]\n", mode, _cleap_stop_timer() );
	//!Unbind Texture
	hipUnbindTexture(tex_triangles);
	// unmap buffer object
	hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);
	hipHostFree(h_listo);

	return *flips;

}
CLEAP_RESULT cleap_clear_mesh(_cleap_mesh *m){

	if(m->status){
		free(m->vnc_data.v);
		free(m->vnc_data.n);
		free(m->vnc_data.c);
		free(m->edge_data.n);
		free(m->edge_data.a);
		free(m->edge_data.b);
		free(m->edge_data.op);
		free(m->triangles);

		if(m->dm->status){
			hipFree(m->dm->d_edges_n);
			hipFree(m->dm->d_edges_a);
			hipFree(m->dm->d_edges_b);
			hipFree(m->dm->d_edges_op);

			hipFree(m->dm->d_trirel);
			hipFree(m->dm->d_trireservs);
			hipFree(m->dm->d_listo);

			glDeleteBuffers(1, &m->dm->vbo_v );
			glDeleteBuffers(1, &m->dm->vbo_n );
			glDeleteBuffers(1, &m->dm->vbo_c );
			glDeleteBuffers(1, &m->dm->eab );

			// opengl method above should have deleted the arrays, this following lines are the equivalent on cuda
			float4 *d_vbo_v, *d_vbo_n, *d_vbo_c;
			GLuint *d_eab;
			size_t bytes=0;

			hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
			hipGraphicsMapResources(1, &m->dm->vbo_n_cuda, 0);
			hipGraphicsMapResources(1, &m->dm->vbo_c_cuda, 0);
			hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);
			hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &bytes, m->dm->vbo_v_cuda);
			hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_n, &bytes, m->dm->vbo_n_cuda);
			hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_c, &bytes, m->dm->vbo_c_cuda);
			hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &bytes, m->dm->eab_cuda);

			hipFree(d_vbo_v);
			hipFree(d_vbo_n);
			hipFree(d_vbo_c);
			hipFree(d_eab);

		}
		delete m->dm;
		delete m;
	}
	//printf("CLEAP::clear_mesh::");
	//_cleap_print_gpu_mem();
	return CLEAP_SUCCESS;
}

CLEAP_RESULT cleap_save_mesh(_cleap_mesh *m, const char *filename){

	//before saving mesh, we have to sync the data from device and host
	cleap_sync_mesh(m);
	int vcount = cleap_get_vertex_count(m);
	int fcount = cleap_get_face_count(m);
	int ecount = cleap_get_edge_count(m);
	//following line is for computer with other languages.
	setlocale(LC_NUMERIC, "POSIX");
	FILE *file_descriptor = fopen(filename,"w");
	fprintf(file_descriptor,"OFF\n");
	fprintf(file_descriptor,"%d %d %d\n",vcount, fcount, ecount);
	for(int i=0; i<vcount; i++) {
		fprintf(file_descriptor,"%f %f %f\n",m->vnc_data.v[i].x,m->vnc_data.v[i].y,m->vnc_data.v[i].z);
	}
	for(int i=0; i<fcount; i++) {
		fprintf(file_descriptor,"%d %d %d %d\n", 3, m->triangles[i*3+0],m->triangles[i*3+1], m->triangles[i*3+2] );
	}
	fclose(file_descriptor);
	setlocale(LC_NUMERIC, "");
	return CLEAP_SUCCESS;
}

CLEAP_RESULT cleap_save_mesh_no_sync(_cleap_mesh *m, const char *filename){

	int vcount = cleap_get_vertex_count(m);
	int fcount = cleap_get_face_count(m);
	int ecount = cleap_get_edge_count(m);
	//following line is for computer with other languages.
	setlocale(LC_NUMERIC, "POSIX");
	FILE *file_descriptor = fopen(filename,"w");
	fprintf(file_descriptor,"OFF\n");
	fprintf(file_descriptor,"%d %d %d\n",vcount, fcount, ecount);
	for(int i=0; i<vcount; i++) {
		fprintf(file_descriptor,"%f %f %f\n",m->vnc_data.v[i].x,m->vnc_data.v[i].y,m->vnc_data.v[i].z);
	}
	for(int i=0; i<fcount; i++) {
		fprintf(file_descriptor,"%d %d %d %d\n", 3, m->triangles[i*3+0],m->triangles[i*3+1], m->triangles[i*3+2] );
	}
	fclose(file_descriptor);
	setlocale(LC_NUMERIC, "");
	return CLEAP_SUCCESS;
}

void _cleap_start_timer(){
    gettimeofday(&t_ini, NULL); //Tiempo de Inicio
}
double _cleap_stop_timer(){
    gettimeofday(&t_fin, NULL); //Tiempo de Termino
    return (double)(t_fin.tv_sec + (double)t_fin.tv_usec/1000000) - (double)(t_ini.tv_sec + (double)t_ini.tv_usec/1000000);
}

void _cleap_reset_minmax(_cleap_mesh* m){

	m->min_coords.x = FLT_MAX;
	m->min_coords.y = FLT_MAX;
	m->min_coords.z = FLT_MAX;
	m->max_coords.x = -1*FLT_MAX;
	m->max_coords.y = -1*FLT_MAX;
	m->max_coords.z = -1*FLT_MAX;
}

CLEAP_RESULT _cleap_normalize_normals(_cleap_mesh *m){

	//printf("CLEAP::kernel::normalize_normals::");
	size_t bytes;
	float4 *dptr;
	int vcount = cleap_get_vertex_count(m);
	cleap_device_mesh *dm = m->dm;
	hipGraphicsMapResources(1, &dm->vbo_n_cuda, 0);
	hipGraphicsResourceGetMappedPointer((void**)&dptr, &bytes, dm->vbo_n_cuda );

	dim3 dimBlock(CLEAP_CUDA_BLOCKSIZE);
	dim3 dimGrid( (vcount+CLEAP_CUDA_BLOCKSIZE)/dimBlock.x);
	hipDeviceSynchronize();
	cleap_kernel_normalize_normals<<< dimGrid, dimBlock >>>(dptr, vcount);
	hipDeviceSynchronize();
	// unmap buffer object
	hipGraphicsUnmapResources(1, &dm->vbo_n_cuda, 0);
	//printf("ok\n");

	return CLEAP_SUCCESS;
}


CLEAP_RESULT _cleap_device_load_mesh(_cleap_mesh* m){


	// CLEAP::DEVICE_LOAD:: create instance of device_mesh struct
	m->dm = new cleap_device_mesh();
	cleap_device_mesh *dmesh = m->dm;
	hipError_t err;
	// CLEAP::DEVICE_LOAD:: get sizes of _cleap_mesh arrays, in bytes
	GLintptr size = cleap_get_vertex_count(m) *4* sizeof(float);
	GLintptr triangles_bytes_size = sizeof(GLuint)*cleap_get_face_count(m)*3;

	// CLEAP::DEVICE_LOAD:: vbo vertex data
	glGenBuffers(1, &dmesh->vbo_v);
	glBindBuffer(GL_ARRAY_BUFFER, dmesh->vbo_v);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBufferSubData(GL_ARRAY_BUFFER, 0, size, m->vnc_data.v);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	err = hipGraphicsGLRegisterBuffer(&dmesh->vbo_v_cuda, dmesh->vbo_v, cudaGraphicsMapFlagsNone);
	if( err != hipSuccess )
		printf("CLEAP::device_load_mesh::cudaGraphicsRegisterBuffer::vbo_p:: %s\n", hipGetErrorString(err));
	
	// CLEAP::DEVICE_LOAD:: vbo normal data
	glGenBuffers(1, &dmesh->vbo_n);
	glBindBuffer(GL_ARRAY_BUFFER, dmesh->vbo_n);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBufferSubData(GL_ARRAY_BUFFER, 0, size, m->vnc_data.n);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	err = hipGraphicsGLRegisterBuffer(&dmesh->vbo_n_cuda, dmesh->vbo_n, cudaGraphicsMapFlagsNone);
	if( err != hipSuccess )	
		printf("CLEAP::device_load_mesh::cudaGraphicsRegisterBuffer::vbo_n:: %s\n", hipGetErrorString(err));

	// CLEAP::DEVICE_LOAD:: vbo color data
	glGenBuffers(1, &dmesh->vbo_c);
	glBindBuffer(GL_ARRAY_BUFFER, dmesh->vbo_c);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBufferSubData(GL_ARRAY_BUFFER, 0, size, m->vnc_data.c);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	err = hipGraphicsGLRegisterBuffer(&dmesh->vbo_c_cuda, dmesh->vbo_c, cudaGraphicsMapFlagsNone);
	if( err != hipSuccess )
		printf("CLEAP::device_load_mesh::cudaGraphicsRegisterBuffer::vbo_c:: %s\n", hipGetErrorString(err));

	// CLEAP::DEVICE_LOAD:: eab data
	glGenBuffers(1, &dmesh->eab);                                                                                   // Generate buffer //index VBO
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, dmesh->eab);                                                             // Bind the element array buffer
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, triangles_bytes_size , 0, GL_STATIC_DRAW);
	glBufferSubData(GL_ELEMENT_ARRAY_BUFFER, 0, triangles_bytes_size, m->triangles);                                 //llenar indices por OpenGL -- OPCION A
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
	err = hipGraphicsGLRegisterBuffer(&dmesh->eab_cuda, dmesh->eab, cudaGraphicsMapFlagsNone);
	if( err != hipSuccess )
		printf("CLEAP::device_load_mesh::cudaGraphicsRegisterBuffer::eab:: %s\n", hipGetErrorString(err));

	// CLEAP::DEVICE_LOAD:: edges data
	// CLEAP::DEVICE_LOAD:: malloc mesh and aux arrays
	size_t edge_bytes_size  = sizeof(int2)* cleap_get_edge_count(m);
	size_t face_bytes_size = sizeof(int)*cleap_get_face_count(m);
	hipMalloc( (void**) &dmesh->d_edges_n , edge_bytes_size );
	hipMalloc( (void**) &dmesh->d_edges_a , edge_bytes_size );
	hipMalloc( (void**) &dmesh->d_edges_b , edge_bytes_size );
	hipMalloc( (void**) &dmesh->d_edges_op , edge_bytes_size );
	hipMalloc( (void**) &dmesh->d_trirel, face_bytes_size );
	hipMalloc( (void**) &dmesh->d_trireservs, face_bytes_size );

	// CLEAP::DEVICE_LOAD:: memcpy mesh and aux arrays
	hipMemcpy( dmesh->d_edges_n, m->edge_data.n , edge_bytes_size, hipMemcpyHostToDevice );
	hipMemcpy( dmesh->d_edges_a, m->edge_data.a , edge_bytes_size, hipMemcpyHostToDevice );
	hipMemcpy( dmesh->d_edges_b, m->edge_data.b , edge_bytes_size, hipMemcpyHostToDevice );
	hipMemcpy( dmesh->d_edges_op, m->edge_data.op , edge_bytes_size, hipMemcpyHostToDevice );

	// CLEAP::DEVICE_LOAD:: add new device mesh entry into the array of device meshes
	// CLEAP::DEVICE_LOAD:: link main mesh with device_mesh id;
	dmesh->status = CLEAP_SUCCESS;
	//printf("CLEAP::device_load_mesh::ok\n");
	//printf("\n");
	fflush(stdout);

	// CLEAP::DEVICE_LOAD:: paint mesh (green by default)
	cleap_paint_mesh(m, 0.0f, 1.0f, 0.0f, 1.0f );

	// CLEAP::DEVICE_LOAD:: normalize normals
	_cleap_normalize_normals(m);

	// CLEAP::DEVICE_LOAD:: print gpu memory
	//printf("CLEAP::");
	//_cleap_print_gpu_mem();

	return CLEAP_SUCCESS;
}


void _cleap_init_array_int(int* h_array, int size, int value){

	int *d_array;
	hipMalloc( (void**) &d_array , size*sizeof(int));
	dim3 dimBlock(CLEAP_CUDA_BLOCKSIZE);
	dim3 dimGrid((size+CLEAP_CUDA_BLOCKSIZE) / dimBlock.x);
	hipDeviceSynchronize();
	cleap_kernel_init_array_int<<< dimGrid, dimBlock >>>(d_array, size, value);
	hipDeviceSynchronize();
	//copy results to host
	hipMemcpy( h_array, d_array, size*sizeof(int), hipMemcpyDeviceToHost );
	hipFree(d_array);
}




void _cleap_init_device_array_int(int* d_array, int length, int value){

	dim3 dimBlock(CLEAP_CUDA_BLOCKSIZE);
	dim3 dimGrid((length+CLEAP_CUDA_BLOCKSIZE) / dimBlock.x);
	hipDeviceSynchronize();
	cleap_kernel_init_array_int<<< dimGrid, dimBlock >>>(d_array, length, value);
	hipDeviceSynchronize();
}

void _cleap_init_device_dual_arrays_int(int* d_array1, int* d_array2, int length, int value, dim3 &dimBlock, dim3 &dimGrid){
	cleap_kernel_init_device_arrays_dual<<< dimGrid, dimBlock >>>(d_array1, d_array2, length, value);
}

void _cleap_print_gpu_mem(){
	size_t free=0, total=0;
	hipMemGetInfo(&free, &total);
	printf("gpu_memory_used::%iMB (%i%%)\n" , (int)((total - free)/(1024*1024)), (int)((float)(total - free)/((float)total)*100.0));
}

int _cleap_choose_best_gpu_id(){

	int num_devices, device, max_device = 0;
	hipGetDeviceCount(&num_devices);
	if (num_devices > 1) {
	      int max_multiprocessors = 0;
	      for (device = 0; device < num_devices; device++) {
		      hipDeviceProp_t properties;
		      hipGetDeviceProperties(&properties, device);
		      if (max_multiprocessors < properties.multiProcessorCount) {
		              max_multiprocessors = properties.multiProcessorCount;
		              max_device = device;
		      }
	      }
	}
	return max_device;
}

void _cleap_print_splash(){

	printf("\n\n************************************************\n");
	printf("****************** cleap-%d.%d.%d *****************\n", CLEAP_VERSION_MAJOR, CLEAP_VERSION_MINOR, CLEAP_VERSION_PATCH);
	printf("************************************************\n");
	printf("			by %s\n\n\n", CLEAP_AUTHOR);
	fflush(stdout);
}

void _cleap_init_cuda(){

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	hipError_t err = cudaGLSetGLDevice( _cleap_choose_best_gpu_id() );
	//printf("CLEAP::init::CudaGLSetGLDevice::%s\n", hipGetErrorString(err));
	//printf("CLEAP::init::gpu::%s\n", deviceProp.name );
	//printf("CLEAP::init::"); _cleap_print_gpu_mem();
	//printf("\n");
}

CLEAP_RESULT _cleap_init_glew(){
	if( glewInit() != GLEW_OK ){
		printf( ">> CLEAP::Init::GLEW Cannot Init\n");
		return CLEAP_FAILURE;
	}
	return CLEAP_SUCCESS;
}
